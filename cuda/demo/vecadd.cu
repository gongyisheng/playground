
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1000000000
#define BLOCK_SIZE 256

__global__ void vector_add(float *out, float *a, float *b, long n) {
    for(long i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    // Initialize arrays
    for (long i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    // Note: The `cudaMalloc` function is defined to take a `void**` as its first argument. 
    // This is because `cudaMalloc` can allocate memory for any data type, not just float. 
    // Using `void**` makes the function more generic.
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Launch kernel
    vector_add<<<1, 1>>>(d_out, d_a, d_b, N);

    // Transfer result back to host
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verify the result
    for (long i = 0; i < N; i++) {
        if (out[i] != 3.0f) {
            printf("Error: out[%ld] = %f\n", i, out[i]);
            break;
        }
    }

    // Cleanup
    free(a); 
    free(b); 
    free(out);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_out);

    printf("Vector addition completed successfully\n");

    return 0;
}