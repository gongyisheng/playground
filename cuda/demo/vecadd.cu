
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000000
#define BLOCK_SIZE 256

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Determine grid and block sizes
    int blockSize = BLOCK_SIZE;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Launch kernel
    vector_add<<<numBlocks, blockSize>>>(d_out, d_a, d_b, N);

    // Transfer result back to host
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (out[i] != 3.0f) {
            printf("Error: out[%d] = %f\n", i, out[i]);
            break;
        }
    }

    // Cleanup
    free(a); 
    free(b); 
    free(out);
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_out);

    printf("Vector addition completed successfully\n");

    return 0;
}